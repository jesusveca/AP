#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cassert>
#include <cmath>
#include <stdio.h>
#include <iostream>
#include <iomanip>

using namespace std;
using namespace cv;

uchar4        	*device_RGBA_Imagen__;
unsigned char 	*device_GRIS_Imagen__;

Mat imageInputRGBA;
Mat imageOutputRGBA;

uchar4 *d_inputImageRGBA__;
uchar4 *d_outputImageRGBA__;
        
float *h_filter__;

size_t numRows() { return imageInputRGBA.rows; }
size_t numCols() { return imageInputRGBA.cols; }


__global__
void gaussian_blur(const unsigned char* const in,	unsigned char* const out,	int numRows, int numCols, const float* const filter, const int filterWidth){
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(Col < numRows && Row< numRows){
		int pixVal =0;
		int pixels =0;
		for (int blurRow = -filterWidth; blurRow<filterWidth+1; ++blurRow){
			for(int blurCol = -filterWidth; blurCol< filterWidth+1; ++blurCol){
				int curRow= Row+blurRow;
				int curCol= Col+blurCol;
				if(curRow>-1 && curRow<numRows && curCol>-1 && curCol<numCols){
					pixVal+=in[curRow*numRows + curCol];
					pixels++;
				}
			}
		}
		out[Row*numRows+Col]=(unsigned char)(pixVal/pixels);
	}
    __syncthreads();

}

__global__
void separateChannels(const uchar4* const inputImageRGBA, int numRows, int numCols, unsigned char* const redChannel, unsigned char* const greenChannel, unsigned char* const blueChannel){

    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = idx_y * numCols + idx_x;

    if (idx_x < numCols && idx_y < numRows){
        uchar4 input = inputImageRGBA[idx];
        __syncthreads();
        redChannel[idx] = input.x;
        greenChannel[idx] = input.y;
        blueChannel[idx] = input.z;
    }
}

__global__
void recombineChannels(const unsigned char* const redChannel, const unsigned char* const greenChannel, const unsigned char* const blueChannel, uchar4* const outputImageRGBA, int numRows, int numCols) {
    const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

    //make sure we don't try and access memory outside the image
    //by having any threads mapped there return early
    if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
        return;

    unsigned char red   = redChannel[thread_1D_pos];
    unsigned char green = greenChannel[thread_1D_pos];
    unsigned char blue  = blueChannel[thread_1D_pos];

    //Alpha should be 255 for no transparency
    uchar4 outputPixel = make_uchar4(red, green, blue, 255);

    outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  //original
  hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage);
  hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage);
  hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage);

  //TODO:
  //Allocate memory for the filter on the GPU
  //Use the pointer d_filter that we have already declared for you
  //You need to allocate memory for the filter with hipMalloc
  //be sure to use checkCudaErrors like the above examples to
  //be able to tell if anything goes wrong
  //IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc
  hipMalloc(&d_filter, sizeof(float) * filterWidth * filterWidth);

  //TODO:
  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!
  hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice);
}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA, uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols, unsigned char *d_redBlurred, unsigned char *d_greenBlurred, unsigned char *d_blueBlurred, const int filterWidth){
    //TODO: Set reasonable block size (i.e., number of threads per block)
    const dim3 blockSize(32, 32);

    //TODO:
    //Compute correct grid size (i.e., number of blocks per kernel launch)
    //from the image size and and block size.
    const dim3 gridSize( (numCols + blockSize.x - 1) / blockSize.x, (numRows + blockSize.y - 1) / blockSize.y);

    //TODO: Launch a kernel for separating the RGBA image into different color channels
    separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);

    // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
    // launching your kernel to make sure that you didn't make any mistakes.
    hipDeviceSynchronize(); 
	
    //TODO: Call your convolution kernel here 3 times, once for each color channel.
    gaussian_blur<<<gridSize, blockSize>>>(  d_red,   d_redBlurred, numRows, numCols, d_filter, filterWidth);
    gaussian_blur<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
    gaussian_blur<<<gridSize, blockSize>>>( d_blue,  d_blueBlurred, numRows, numCols, d_filter, filterWidth);

    // Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
    // launching your kernel to make sure that you didn't make any mistakes.
    hipDeviceSynchronize(); 


    // Now we recombine your results. We take care of launching this kernel for you.
    //
    // NOTE: This kernel launch depends on the gridSize and blockSize variables,
    // which you must set yourself.
    recombineChannels<<<gridSize, blockSize>>>(d_redBlurred, d_greenBlurred, d_blueBlurred, d_outputImageRGBA, numRows, numCols);
    hipDeviceSynchronize(); 
}

void preProcess(uchar4 **h_inputImageRGBA, uchar4 **h_outputImageRGBA,
                uchar4 **d_inputImageRGBA, uchar4 **d_outputImageRGBA,
                unsigned char **d_redBlurred,
                unsigned char **d_greenBlurred,
                unsigned char **d_blueBlurred,
                float **h_filter, int *filterWidth,
                const std::string &filename) {

    cv::Mat image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
    cv::cvtColor(image, imageInputRGBA, CV_BGR2RGBA);
    imageOutputRGBA.create(image.rows, image.cols, CV_8UC4);

    *h_inputImageRGBA  = (uchar4 *)imageInputRGBA.ptr<unsigned char>(0);
    *h_outputImageRGBA = (uchar4 *)imageOutputRGBA.ptr<unsigned char>(0);

    const size_t numPixels = numRows() * numCols();
    //allocate memory on the device for both input and output
    hipMalloc(d_inputImageRGBA, sizeof(uchar4) * numPixels);
	hipMalloc(d_outputImageRGBA, sizeof(uchar4) * numPixels);
	hipMemset(*d_outputImageRGBA, 0, numPixels * sizeof(uchar4));

    hipMemcpy(*d_inputImageRGBA, *h_inputImageRGBA, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice);

    d_inputImageRGBA__  = *d_inputImageRGBA;
    d_outputImageRGBA__ = *d_outputImageRGBA;

    //now create the filter that they will use
    const int blurKernelWidth = 9;
    const float blurKernelSigma = 2.;

    *filterWidth = blurKernelWidth;

    //create and fill the filter we will convolve with
    *h_filter = new float[blurKernelWidth * blurKernelWidth];
    h_filter__ = *h_filter;

    float filterSum = 0.f; //for normalization

    for (int r = -blurKernelWidth/2; r <= blurKernelWidth/2; ++r) {
        for (int c = -blurKernelWidth/2; c <= blurKernelWidth/2; ++c) {
            float filterValue = expf( -(float)(c * c + r * r) / (2.f * blurKernelSigma * blurKernelSigma));
            (*h_filter)[(r + blurKernelWidth/2) * blurKernelWidth + c + blurKernelWidth/2] = filterValue;
            filterSum += filterValue;
        }
    }

    float normalizationFactor = 1.f / filterSum;

    for (int r = -blurKernelWidth/2; r <= blurKernelWidth/2; ++r) {
        for (int c = -blurKernelWidth/2; c <= blurKernelWidth/2; ++c) {
            (*h_filter)[(r + blurKernelWidth/2) * blurKernelWidth + c + blurKernelWidth/2] *= normalizationFactor;
        }
    }

    //blurred
	hipMalloc(d_redBlurred,    sizeof(unsigned char) * numPixels);
    hipMalloc(d_greenBlurred,  sizeof(unsigned char) * numPixels);
    hipMalloc(d_blueBlurred,   sizeof(unsigned char) * numPixels);
    hipMemset(*d_redBlurred,   0, sizeof(unsigned char) * numPixels);
    hipMemset(*d_greenBlurred, 0, sizeof(unsigned char) * numPixels);
    hipMemset(*d_blueBlurred,  0, sizeof(unsigned char) * numPixels);
}

void postProcess(const std::string& output_file) {
    const int numPixels = numRows() * numCols();
    //copy the output back to the host
    hipMemcpy(imageOutputRGBA.ptr<unsigned char>(0), d_outputImageRGBA__, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost);

    cv::Mat imageOutputBGR;
    cv::cvtColor(imageOutputRGBA, imageOutputBGR, CV_RGBA2BGR);
    //output the image
    cv::imwrite(output_file.c_str(), imageOutputBGR);

    //cleanup
    hipFree(d_inputImageRGBA__);
    hipFree(d_outputImageRGBA__);
    delete[] h_filter__;
}

int main(int argc, char **argv) {

	uchar4 *h_inputImageRGBA,  *d_inputImageRGBA;
    uchar4 *h_outputImageRGBA, *d_outputImageRGBA;

    unsigned char *d_redBlurred, *d_greenBlurred, *d_blueBlurred;

    float *h_filter;
    int    filterWidth;


	std::string input_file;
	input_file="lena.jpg";

    std::string output_file;
	output_file="SALIDA.jpg";



	preProcess(&h_inputImageRGBA, &h_outputImageRGBA, &d_inputImageRGBA, &d_outputImageRGBA, &d_redBlurred, &d_greenBlurred, &d_blueBlurred, &h_filter, &filterWidth, input_file);


	allocateMemoryAndCopyToGPU(numRows(), numCols(), h_filter, filterWidth);

	your_gaussian_blur(h_inputImageRGBA, d_inputImageRGBA, d_outputImageRGBA, numRows(), numCols(), d_redBlurred, d_greenBlurred, d_blueBlurred, filterWidth);
    hipDeviceSynchronize(); 
    postProcess(output_file);
    return 0;
}