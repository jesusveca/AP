
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <time.h>



#include <stdio.h>
#include <stdlib.h>

#define N 20
#define BLOCK_DIM 10

using namespace std;
#define TILE_WIDTH 4


/*
// EJERCICIO DE LA SECCION 4.7
__global__
void matMultKernel_tile_siete_siente(int *d_M, int *d_N, int *d_P, int Width){

  extern __shared__ int Mds[][];
  extern __shared__ int Nds[][];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
 
  int Row = by*TILE_WIDTH + ty;
  int Col = bx*TILE_WIDTH + tx;

  float Pvalue = 0;
  int  ph,k;
  for(ph = 0; ph < Width/TILE_WIDTH; ++ph){
    // Collaborative loading of M and N tiles into shared memory
    if ( (Row<Width) && (ph*TILE_WIDTH+tx)<Width )
        Mds[ty][tx] = d_M[Row*Width + ph*TILE_WIDTH + tx];
    if ( (ph*TILE_WIDTH+ty)<Width && Col<Width )
        Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty) + Col];
    
    __syncthreads();
    for(k = 0; k < TILE_WIDTH; ++k){
      Pvalue += Mds[ty][k] * Nds[k][tx];
     }
     __syncthreads();
  }
  if ( (Row<Width) && (Col<Width))
    d_P[Row*Width + Col] = Pvalue;
}

*/

// EJERCICIO DE LA SECCION 4.6
__global__
void matMultKernel_tile_seis(int *d_M, int *d_N, int *d_P, int Width){

  __shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ int Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
 
  int Row = by*TILE_WIDTH + ty;
  int Col = bx*TILE_WIDTH + tx;

  float Pvalue = 0;
  int  ph,k;
  for(ph = 0; ph < Width/TILE_WIDTH; ++ph){
    // Collaborative loading of M and N tiles into shared memory
    if ( (Row<Width) && (ph*TILE_WIDTH+tx)<Width )
        Mds[ty][tx] = d_M[Row*Width + ph*TILE_WIDTH + tx];
    if ( (ph*TILE_WIDTH+ty)<Width && Col<Width )
        Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty) + Col];
    
    __syncthreads();
    for(k = 0; k < TILE_WIDTH; ++k){
      Pvalue += Mds[ty][k] * Nds[k][tx];
     }
     __syncthreads();
  }
  if ( (Row<Width) && (Col<Width))
    d_P[Row*Width + Col] = Pvalue;
}



// EJERCICIO DE LA SECCION 4.4
__global__
void matMultKernel_tile(int *d_M, int *d_N, int *d_P, int Width){

  __shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ int Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
 
  int Row = by*TILE_WIDTH + ty;
  int Col = bx*TILE_WIDTH + tx;

  float Pvalue = 0;
  int  ph,k;
  for(ph = 0; ph < Width/TILE_WIDTH; ++ph){
     Mds[ty][tx] = d_M[Row*Width+ph*TILE_WIDTH + tx];
     Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty) * Width + Col];
     __syncthreads();
     for(k = 0; k < TILE_WIDTH; ++k){
      Pvalue += Mds[ty][k] * Nds[k][tx];
     }
     __syncthreads();
  }
  d_P[Row*Width + Col] = Pvalue;
}

// EJERCICIO DE LA SECCION 4.2

__global__
void matMultKernel(int *d_M, int *d_N, int *d_P, int Width){
  int Row = blockIdx.y*blockDim.y + threadIdx.y;
  int Col = blockIdx.x*blockDim.x + threadIdx.x;
  int k = 0;
  if(Row < Width && Col < Width){
      float Pvalue = 0;
      for(k = 0; k < Width; ++k){
          Pvalue += d_M[Row*Width + k] * d_N[k*Width+Col];
      }
      d_P[Row*Width+Col] = Pvalue;
  }
}

__global__ 
void sum_Matrices_Normal (int *a, int *b, int *c) {
    int columna = blockIdx.x * blockDim.x + threadIdx.x;
    int fila = blockIdx.y * blockDim.y + threadIdx.y;
    int id = columna + fila * N;
    if (columna < N && fila < N) {
        c[id] = a[id] + b[id];
    }

}

void imprimir_Matriz(int matrix[N][N]){
    for(int i=0;i<N;i++){
        for(int j=0; j<N; j++){
            cout<<matrix[i][j]<<' ';
        }
        cout<<endl;
    }
}

void inicio_matrices(){
    int a[N][N], b[N][N], c[N][N];
    int *dev_a, *dev_b, *dev_c;

    int size = N * N * sizeof(int);
    srand(time(NULL));
    for(int i=0; i<N; i++)
        for (int j=0; j<N; j++){
            a[i][j] = 1;
            b[i][j] = 1;
        }

    imprimir_Matriz(a);
    cout<<endl;
    imprimir_Matriz(b);


    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    //dim3 dimBlock(N,N); // cuantos threads se ejecutaran juntos y que compartiran memoria en un sigle proccessor
    //dim3 dimGrid(1,1); // un grupo de thread block que se ejecutan en un sigle cuda program logically in parallel
    
    //dim3 dimGrid(ceil(N/1024.0),ceil(N/1024.0),1);
    //dim3 dimBlock(1024,1024,1);  /*Dimensiónde 2 X 2, eje Z desactivado con 1*/
    
    
    dim3 dimGrid(ceil(N/4.0),ceil(N/4.0),1);
    dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);
    matMultKernel_tile_seis<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c, N);
    
    
    //for siete
    //size_t size=  calculate_appropriate_SM_usage(dev_prop.sharedMemPerBlock,...);
    //matMultKernel_tile_siete_siente<<<dimGrid, dimBlock, size>>>(Md, Nd, Pd, Width);
    
    
    
    hipDeviceSynchronize();
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    
    cout<<endl;
    for(int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            printf("%d ", c[i][j] );
        }
        printf("\n");
    }

    hipFree(dev_a);    hipFree(dev_b);    hipFree(dev_c);
}

int main() {
    inicio_matrices();

    return 0;
}