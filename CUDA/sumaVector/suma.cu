
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>

__global__
void vecAddKernel(float *A, float *B, float *C, int n){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < n) C[i] = A[i] + B[i];
}

void vecAdd(float* A, float* B, float* C, int n){
  int size = n*sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_C, size);

  vecAddKernel<<<ceil(n/1024.0), 1024>>>(d_A,d_B,d_C,n);
  
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main(){
  unsigned t0,t1;
  int n,i;
  float *h_A,*h_B,*h_C;
  
  n=1000;
  h_A = (float*) malloc(n*sizeof(float));
  h_B = (float*) malloc(n*sizeof(float));
  h_C = (float*) malloc(n*sizeof(float));
  for(i = 0; i < n; i++){
    h_A[i] = i;
  }
  for(i = 0; i < n; i++){
    h_B[i] = i;
  }
  t0=clock();
  vecAdd(h_A,h_B,h_C,n);
  t1=clock();
  double time = (double(t1-t0)/CLOCKS_PER_SEC);
  for(i = 0; i < n; i++){
    printf("%f ", h_C[i]);
  }
  printf("Finalizado : %f\n: ",time);
  printf("\n");
  return 0;
}
