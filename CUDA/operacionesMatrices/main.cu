
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <time.h>

#include <stdio.h>
#include <stdlib.h>

#define N 5
#define BLOCK_DIM 10
using namespace std;

__global__ 
void adicion_Matrices (int *a, int *b, int *c) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int fila = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + fila * N;
    if (col < N && fila < N) {
        c[index] = a[index] + b[index];
    }
}

__global__ 
void matrix_vector (int *a, int *b, int *c) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < N) {
        for(int i=0;i<N;i++){
            sum += b[i]*a[(i*N)+col];
        }
        c[col] = sum;
    }
}

__global__ 
void adicion_matrices_Filas (int *a, int *b, int *c) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int fila = blockIdx.y * blockDim.y + threadIdx.y;
    for(int i=col; i<N; i++){
        int index = i + fila * N;
        c[index] = a[index] + b[index];
    }
}

__global__ 
void adicion_matrices_Columnas (int *a, int *b, int *c) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int fila = blockIdx.y * blockDim.y + threadIdx.y;
    for(int i=fila; i<N; i++){
        int index = col + i * N;
        c[index] = a[index] + b[index];
    }
}

void imprimir(int matrix[N][N]){
    for(int i=0;i<N;i++){
        for(int j=0; j<N; j++){
            std::cout << matrix[i][j] << '\t';
        }
        std::cout << std::endl;
    }
}

void print_vector(int vector[N]){
    for(int j=0; j<N; j++){
        std::cout << vector[j] << '\t';
    }
}

void adicion_matrix_to_Kernel(){
    int a[N][N], b[N][N], c[N][N];
    int *dev_a, *dev_b, *dev_c;

    int size = N * N * sizeof(int);
    srand(time(NULL));
    for(int i=0; i<N; i++)
        for (int j=0; j<N; j++){
            a[i][j] = rand() % 3;
            b[i][j] = rand() % 3;
        }

    imprimir(a);
    std::cout << std::endl;
    imprimir(b);
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);

    adicion_Matrices<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c);
    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    std::cout << std::endl;

    imprimir (c);

    /*
    for(int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            printf("%d\t", c[i][j] );
        }
        printf("\n");
    }
    */
}

int main() {
    int a[N][N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    int size = N * N * sizeof(int);
    srand(time(NULL));
    for(int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            a[i][j] = rand() % 9;
        }
        b[i] = rand() % 9;
    }

    imprimir(a);
    std::cout << std::endl;
    print_vector(b);

    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N* sizeof(int));

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    matrix_vector<<<N/256+1,256>>>(dev_a,dev_b,dev_c);

    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    std::cout << std::endl;
    print_vector(c);

    return 0;
}