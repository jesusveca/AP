#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ 
void adicion_vectores(float x[], float y[], float z[], int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) 
    z[idx] = x[idx] + y[idx];
}  

int main(int argc, char* argv[]) {
   int n, i;
   float *host_x, *host_y, *host_z, *device_x, *device_y, *device_z;
   int threads_bloque, tamanoBloque;
   size_t size;
   n=10;
   size = n*sizeof(float);
   host_x = (float*) malloc(size);      host_y = (float*) malloc(size);     host_z = (float*) malloc(size);

   for (i = 0; i < n; i++) {
      host_x[i] = i+10;      host_y[i] = n-3;
   }

   hipMalloc(&device_x, size);
   hipMalloc(&device_y, size);
   hipMalloc(&device_z, size);

   hipMemcpy(device_x, host_x, size, hipMemcpyHostToDevice);
   hipMemcpy(device_y, host_y, size, hipMemcpyHostToDevice);

   threads_bloque = 256;

   tamanoBloque = (n + threads_bloque - 1)/threads_bloque;
   adicion_vectores<<<tamanoBloque, threads_bloque>>>(device_x, device_y, device_z, n);

   hipDeviceSynchronize();

   hipMemcpy(host_z, device_z, size, hipMemcpyDeviceToHost);
   printf("La suma es: \n");
   for (i = 0; i < n; i++)
      printf("%.1f ", host_z[i]);
   printf("\n");
   return 0;
}  